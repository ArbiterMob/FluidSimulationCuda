// CUDA first version naive (multiple cells for each thread)

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

#include <hip/hip_runtime.h>

// GPU Constants
__constant__ int N;       // (WidthGrid - 2) 
__constant__ float DT;    // Instantaneous change in time (timestep)
__constant__ float VIS;   // Viscosity coefficient
__constant__ float DIFF;  // Diffusion coefficient

// CPU Global Variables
int hN = (1<<13) - 2;
float hDT = 0.016f;
float hVIS = 0.0025f;
float hDIFF = 0.1f;
int GRID_DIVISION_FACTOR = 4; // explain what this is !!!

// SWAP macro
#define SWAP(x0, x) {float *tmp = x0; x0 = x; x = tmp;}

// Function to check CUDA errors
#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if (error != hipSuccess) \
    { \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
        fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(EXIT_FAILURE); \
    } \
}

// Function to debug a float grid
void printDebug(/*char *string,*/ float *x) {
    int i, j;

    //printf("Debugging -> %s\n", string);

    for (i = 0; i < hN + 2; i++) {
        for (j = 0; j < hN + 2; j++) {
            printf("[%f] ", x[j + i * (hN + 2)]);
        }
        printf("\n");
    }
    printf("\n\n");
}

// Function to debug and print the state of the grid
void printStateGrid(float *dens, float *u, float *v) {
    int i, j;

    printf("---------------------------------------\n");
    printf("DENSITY\n");
    for (i = 0; i < hN + 2; i++) {
        for (j = 0; j < hN + 2; j++) {
            printf("[%f] ", dens[j + i * (hN +2)]);
        }
        printf("\n");
    }
    printf("\n\n");

    printf("VELOCITY\n");
    for (i = 0; i < hN + 2; i++) {
        for (j = 0; j < hN + 2; j++) {
            printf("[%f, %f] ", u[j + i * (hN +2)], v[j + i * (hN +2)]);
        }
        printf("\n");
    }
}

// Function to measure time in seconds
double cpuSecond() {
    struct timespec ts;
    timespec_get(&ts, TIME_UTC);
    return ((double)ts.tv_sec + (double)ts.tv_nsec * 1.e-9);
}

// CUDA device function to set the boundary conditions (borders)
__device__ void set_bndOnGPU(int b, float *d_x, int ix, int iy, int tid) {
    if (ix == 0 && iy < N + 1 && iy > 0)
        d_x[tid] = b == 1 ? -d_x[tid + 1] : d_x[tid + 1];
    else if (ix == N + 1 && iy < N + 1 && iy > 0)
        d_x[tid] = b == 1 ? -d_x[tid - 1] : d_x[tid - 1];
    else if (iy == 0 && ix > 0 && ix < N + 1)
        d_x[tid] = b == 2 ? -d_x[tid + N + 2] : d_x[tid + N + 2];
    else if (iy == N + 1 && ix > 0 && ix < N + 1)
        d_x[tid] = b == 2 ? -d_x[tid - N - 2] : d_x[tid - N - 2];
}   

// CUDA device function to set the boundary conditions (corners)
__device__ void set_crnOnGPU(int b, float *d_x, int ix, int iy, int tid) {
    if (ix == 0 && iy == 0)
        d_x[tid] = 0.5f * (d_x[tid + 1] + d_x[tid + N + 2]);
    else if (ix == 0 && iy == N + 1)
        d_x[tid] = 0.5f * (d_x[tid + 1] + d_x[tid - N - 2]);
    else if (ix == N + 1 && iy == 0)
        d_x[tid] = 0.5f * (d_x[tid - 1] + d_x[tid + N + 2]);
    else if (ix == N + 1 && iy == N + 1) 
        d_x[tid] = 0.5f * (d_x[tid - 1] + d_x[tid - N - 2]);
} 

// CUDA device function to compute borders and corner
__device__ void setBordersOnGPU(int b, float *x, int start_x, int start_y, int section_size_x, int section_size_y) {
    int i, j;

    //printf("start_x %d | start_y %d\nDEBUG - %d\n\n", start_x, start_y, start_y == 0 && start_x == ((blockDim.x - 1) + blockIdx.x * blockDim.x) * section_size_x);

    if (start_x == 0 && start_y == 0) {
        for (i = 0; i < section_size_y; i++) {
            int nIY = i;
            int nTid = i * (N + 2);
            set_bndOnGPU(b, x, 0, nIY, nTid);
        }
        for (j = 0; j < section_size_x; j++) {
            int nIX = j;
            int nTid = nIX;
            set_bndOnGPU(b, x, nIX, 0, nTid);
        }
        set_crnOnGPU(b, x, 0, 0, 0); 
    } else if (start_x == ((blockDim.x - 1) + blockIdx.x * blockDim.x) * section_size_x && start_y == ((blockDim.y - 1) + blockIdx.y * blockDim.y) * section_size_y) {
        for (i = 0; i < section_size_y; i++) {
            int nIY = start_y + i;
            int nTid = (N + 1) + nIY * (N + 2);
            set_bndOnGPU(b, x, N + 1, nIY, nTid);
        }
        for (j = 0; j < section_size_x; j++) {
            int nIX = start_x + j;
            int nTid = nIX + (N + 1) * (N + 2);
            set_bndOnGPU(b, x, nIX, N + 1, nTid);
        }
        set_crnOnGPU(b, x, N + 1, N + 1, (N + 1) + (N + 1) * (N + 2));
    } else if (start_y == 0 && start_x == ((blockDim.x - 1) + blockIdx.x * blockDim.x) * section_size_x) {
        for (j = 0; j < section_size_x; j++) {
            int nIX = start_x + j;
            int nTid = nIX;
            set_bndOnGPU(b, x, nIX, 0, nTid);
        }
        for (i = 0; i < section_size_y; i++) {
            int nIY = start_y + i;
            int nTid = (N + 1) + nIY * (N + 2);
            set_bndOnGPU(b, x, N + 1, nIY, nTid);
        }
        set_crnOnGPU(b, x, N + 1, 0, N + 1);
    } else if (start_x == 0 && start_y == ((blockDim.y - 1) + blockIdx.y * blockDim.y) * section_size_y) {
        for (i = 0; i < section_size_y; i++) {
            int nIY = start_y + i;
            int nTid = nIY * (N + 2);
            set_bndOnGPU(b, x, 0, nIY, nTid);
        }
        for (j = 0; j < section_size_x; j++) {
            int nIX = start_x + j;
            int nTid = nIX + (N + 1) * (N + 2);
            set_bndOnGPU(b, x, nIX, N + 1, nTid);
        }
        set_crnOnGPU(b, x, 0, N + 1, (N + 1) * (N + 2));
    } else if (start_x == 0) {
        for (i = 0; i < section_size_y; i++) {
            int nIY = start_y + i;
            int nTid = nIY * (N + 2);
            set_bndOnGPU(b, x, 0, nIY, nTid);
        }
    } else if (start_y == 0) {
        for (j = 0; j < section_size_x; j++) {
            int nIX = start_x + j;
            int nTid = nIX;
            set_bndOnGPU(b, x, nIX, 0, nTid);
        }
    } else if (start_x == ((blockDim.x - 1) + blockIdx.x * blockDim.x) * section_size_x) {
        for (i = 0; i < section_size_y; i++) {
            int nIY = start_y + i;
            int nTid = (N + 1) + nIY * (N + 2);
            set_bndOnGPU(b, x, N + 1, nIY, nTid);
        }
    } else if (start_y == ((blockDim.y - 1) + blockIdx.y * blockDim.y) * section_size_y) {
        for (j = 0; j < section_size_x; j++) {
            int nIX = start_x + j;
            int nTid = nIX + (N + 1) * (N + 2);
            set_bndOnGPU(b, x, nIX, N + 1, nTid);
        }
    }
}

// CUDA kernel function to  ADD EXTERNAL SOURCES
__global__ void add_sourceOnGPU(float *d_x, float *d_s) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int i, j;

    int size = (N + 2);
    int section_size_x = (size - 1) / (blockDim.x * gridDim.x) + 1;
    int section_size_y = (size - 1) / (blockDim.y * gridDim.y) + 1;

    int start_x = ix * section_size_x;
    int start_y = iy * section_size_y;
    //printf("ix %d | iy %d | section_size_x %d | section_size_y %d | start_x %d | start_y %d\n",
    //    ix, iy, section_size_x, section_size_y, start_x, start_y);

    for (i = 0; i < section_size_y; i++) {
        for (j = 0; j < section_size_x; j++) {
            int nIX = start_x + j;
            int nIY = start_y + i;
            int nTid = nIX + nIY * (N + 2);
            //printf("ix %d | iy %d | nIX %d | nIY %d | nTid %d\n", 
            //    ix, iy, nIX, nIY, nTid);
            if (nIX < (N + 2) && nIY < (N + 2))
                d_x[nTid] += DT * d_s[nTid];
        }
    }
}

// CUDA kernel function to perform DIFFUSION (using Jacobi iteration outside of kernel)
__global__ void diffuseOnGPU(int b, float *d_x, float *d_x0, float *d_xTemp, float alpha, float beta) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int i, j;

    int size = (N + 2);
    int section_size_x = (size - 1) / (blockDim.x * gridDim.x) + 1;
    int section_size_y = (size - 1) / (blockDim.y * gridDim.y) + 1;

    int start_x = ix * section_size_x;
    int start_y = iy * section_size_y;

    for (i = 0; i < section_size_y; i++) {
        for (j = 0; j < section_size_x; j++) {
            int nIX = start_x + j;
            int nIY = start_y + i;
            int nTid = nIX + nIY * (N + 2);
            if (nIX >= 1 && nIX <= N && nIY >= 1 && nIY <= N) {
                d_xTemp[nTid] = (d_x0[nTid] + alpha * (d_x[nTid - 1] + d_x[nTid + 1] +
                    d_x[nTid - N - 2] + d_x[nTid + N + 2])) / beta;
                //printf("d_xTemp id %f in (%d, %d): d_x0[tid] %f | d_x[tid - 1] %f | d_x[tid + 1] %f, d_x[tid - N - 2] %f | d_x[tid + N + 2] %f | beta %f\n",
                //    d_xTemp[nTid], nIX, nIY, d_x0[nTid], d_x[nTid - 1], d_x[nTid + 1], d_x[nTid - N - 2], d_x[nTid + N + 2], beta);
            }
        }
    }
    
    // maybe this is less performant ...
    setBordersOnGPU(b, d_xTemp, start_x, start_y, section_size_x, section_size_y);
}

// CUDA kernel function to perform ADVECTION (using bilinear interpolation)
__global__ void advectOnGPU(int b, float *d_d, float *d_d0, float *d_u, float *d_v) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int i, j;

    int size = (N + 2);
    int section_size_x = (size - 1) / (blockDim.x * gridDim.x) + 1;
    int section_size_y = (size - 1) / (blockDim.y * gridDim.y) + 1;

    int start_x = ix * section_size_x;
    int start_y = iy * section_size_y;

    int i0, j0, i1, j1;
    float x, y, s0, t0, s1, t1, dt0;

    dt0 = DT * N;
    for (i = 0; i < section_size_y; i++) {
        for (j = 0; j < section_size_x; j++) {
            int nIX = start_x + j;
            int nIY = start_y + i;
            int nTid = nIX + nIY * (N + 2);
            if (nIX >= 1 && nIX <= N && nIY >= 1 && nIY <= N) {
                x = nIX - dt0 * d_u[nTid];
                y = nIY - dt0 * d_v[nTid];

                if (x < 0.5) 
                    x = 0.5; 
                if (x > N + 0.5) 
                    x = N + 0.5;
                j0 = (int)x;
                j1 = j0 + 1;

                if (y < 0.5) 
                    y = 0.5; 
                if (y > N + 0.5) 
                    y = N + 0.5; 
                i0 = (int)y;
                i1 = i0 + 1;

                s1 = x - j0;
                s0 = 1 - s1;
                t1 = y - i0;
                t0 = 1 - t1;

                d_d[nTid] = s0 * (t0 * d_d0[j0 + i0 * (N + 2)] + t1 * d_d0[j0 + i1 * (N + 2)]) +
                    s1 * (t0 * d_d0[j1 + i0 * (N + 2)] + t1 * d_d0[j1 + i1 * (N + 2)]);
                //printf("(%f, %f) | d_d[nTid] %f (%d, %d)| d_d0[j0 + i0 * (N + 2)] %f | d_d0[j0 + i1 * (N + 2)] %f | d_d0[j1 + i0 * (N + 2)] %f | d_d0[j1 + i1 * (N + 2)] %f\n",
                //    x, y, d_d[nTid], nIX, nIY, d_d0[j0 + i0 * (N + 2)], d_d0[j0 + i1 * (N + 2)], d_d0[j1 + i0 * (N + 2)], d_d0[j1 + i1 * (N + 2)]);
            }
        }
    }

    // maybe this is less performant ...
    setBordersOnGPU(b, d_d, start_x, start_y, section_size_x, section_size_y);
}

// CUDA kernel function to COMPUTE DIVERGENCE AND PRESSURE
__global__ void computeDivergenceAndPressureOnGPU(float *d_u, float *d_v, float *p, float *div) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int i, j;

    int size = (N + 2);
    int section_size_x = (size - 1) / (blockDim.x * gridDim.x) + 1;
    int section_size_y = (size - 1) / (blockDim.y * gridDim.y) + 1;

    int start_x = ix * section_size_x;
    int start_y = iy * section_size_y;

    float h = 1.0f / N;
    for (i = 0; i < section_size_y; i++) {
        for (j = 0; j < section_size_x; j++) {
            int nIX = start_x + j;
            int nIY = start_y + i;
            int nTid = nIX + nIY * (N + 2);
            if (nIX >= 1 && nIX <= N && nIY >= 1 && nIY <= N) {
                div[nTid] = -0.5f * h * (d_u[nTid + 1] - d_u[nTid - 1] + d_v[nTid + N + 2] - d_v[nTid - N - 2]);
                p[nTid] = 0.0f;
                //printf("div[tid] %f (%d, %d)| d_u[tid + 1] %f | d_u[tid - 1] %f | d_v[tid + N + 2] %f | d_v[tid - N - 2] %f\n",
                //    div[nTid], ix, iy, d_u[nTid + 1], d_u[nTid - 1], d_v[nTid + N + 2], d_v[nTid - N - 2]);
            }
        }
    }

    // maybe this is less performant ...
    setBordersOnGPU(0, div, start_x, start_y, section_size_x, section_size_y);
    setBordersOnGPU(0, p, start_x, start_y, section_size_x, section_size_y);
}

// CUDA kernel to perform the LAST PROJECTION STEP (using Jacobi iteration outside of kernel)
__global__ void lastProjectOnGPU(float *d_u, float *d_v, float *p) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int i, j;

    int size = (N + 2);
    int section_size_x = (size - 1) / (blockDim.x * gridDim.x) + 1;
    int section_size_y = (size - 1) / (blockDim.y * gridDim.y) + 1;

    int start_x = ix * section_size_x;
    int start_y = iy * section_size_y;


    float h = 1.0f / N;
    for (i = 0; i < section_size_y; i++) {
        for (j = 0; j < section_size_x; j++) {
            int nIX = start_x + j;
            int nIY = start_y + i;
            int nTid = nIX + nIY * (N + 2);
            if (nIX >= 1 && nIX <= N && nIY >= 1 && nIY <= N) {
                d_u[nTid] -= 0.5f * (p[nTid + 1] - p[nTid - 1]) / h;
                d_v[nTid] -= 0.5f * (p[nTid + N + 2] - p[nTid - N - 2]) / h;
            }
        }
    }

    // can be done better but it works
    setBordersOnGPU(1, d_u, start_x, start_y, section_size_x, section_size_y);
    setBordersOnGPU(2, d_v, start_x, start_y, section_size_x, section_size_y);

}

// Function to simulate the evolution of density
void dens_step(dim3 grid, dim3 block, float *d_x, float *d_x0, float *d_u, float *d_v, float *d_densTemp) {
    add_sourceOnGPU<<<grid, block>>>(d_x, d_x0);

    float alpha = hDT * hDIFF * hN * hN;
    float beta = 1 + 4 * alpha;
    SWAP(d_x0, d_x);
    for (int k = 0; k < 40; k++) { // inefficient -> multiple kernel calls
        diffuseOnGPU<<<grid, block>>>(0, d_x, d_x0, d_densTemp, alpha, beta);
        SWAP(d_densTemp, d_x);
    }
    
    SWAP(d_x0, d_x);
    advectOnGPU<<<grid, block>>>(0, d_x, d_x0, d_u, d_v);
}

// Function to simulate the evolution of velocity
void vel_step(dim3 grid, dim3 block, float *d_u, float *d_v, float *d_u0, float *d_v0, float *d_uTemp, float *d_vTemp) {
    add_sourceOnGPU<<<grid, block>>>(d_u, d_u0);
    add_sourceOnGPU<<<grid, block>>>(d_v, d_v0);

    SWAP(d_u, d_u0);
    SWAP(d_v, d_v0);
    float alpha = hDT * hVIS * hN * hN;
    float beta = 1 + 4 * alpha;
    for (int k = 0; k < 40; k++) { // inefficient -> multiple kernel calls
        diffuseOnGPU<<<grid, block>>>(1, d_u, d_u0, d_uTemp, alpha, beta);
        diffuseOnGPU<<<grid, block>>>(2, d_v, d_v0, d_vTemp, alpha, beta);
        SWAP(d_uTemp, d_u);
        SWAP(d_vTemp, d_v);
    }
    
    computeDivergenceAndPressureOnGPU<<<grid, block>>>(d_u, d_v, d_u0, d_v0);

    alpha = 1;
    beta = 4;
    // d_u0 is p, d_v0 is div
    for (int k = 0; k < 40; k++) { // inefficient -> multiple kernel calls
        diffuseOnGPU<<<grid, block>>>(0, d_u0, d_v0, d_uTemp, alpha, beta);
        SWAP(d_uTemp, d_u0);
    }
    lastProjectOnGPU<<<grid, block>>>(d_u, d_v, d_u0);

    SWAP(d_u0, d_u);
    SWAP(d_v0, d_v);
    advectOnGPU<<<grid, block>>>(1, d_u, d_u0, d_u0, d_v0);
    advectOnGPU<<<grid, block>>>(2, d_v, d_v0, d_u0, d_v0);

    computeDivergenceAndPressureOnGPU<<<grid, block>>>(d_u, d_v, d_u0, d_v0);
    // d_u0 is p, d_v0 is div
    for (int k = 0; k < 40; k++) { // inefficient -> multiple kernel calls
        diffuseOnGPU<<<grid, block>>>(0, d_u0, d_v0, d_uTemp, alpha, beta);
        SWAP(d_uTemp, d_u0);
    }
    lastProjectOnGPU<<<grid, block>>>(d_u, d_v, d_u0);
}

// Function to initialize the density and velocity
void initializeParameters(float *dens, float *dens_prev, float *u, float *u_prev, float *v, float *v_prev) {
    int i, j;
    int center_x = (hN + 2) / 2, center_y = (hN + 2) / 2;
    int radius = (hN + 2) / 8;

    // density source
    for (i = 0; i < hN + 2; i++) {
        for (j = 0; j < hN + 2; j++) {
            if ((j < center_x + radius) && (j >= center_x - radius) &&
                (i < center_y + radius) && (i >= center_y - radius))
                dens_prev[j + i * (hN + 2)] = (rand() % 100) / 1000.0f;
            else 
                dens_prev[j + i * (hN + 2)] = 0.0f;
            dens[j + i * (hN + 2)] = 0.0f;
        }
    } 

    // velocity
    for (i = 0; i < hN + 2; i++) {
        for (j = 0; j < hN + 2; j++) {
            u_prev[j + i * (hN +2)] = (rand() % 100) / 100.0f;
            v_prev[j + i * (hN +2)] = (rand() % 100) / 100.0f;

            u[j + i * (hN +2)] = 0.0f;
            v[j + i * (hN +2)] = 0.0f;
        }
    }
}

int main(int argc, char **argv) {
     // Check command line arguments
    if (argc != 3) {
        printf("Usage: %s <block_dim_x> <block_dim_y>\n", argv[0]);
        return 1;
    }
    
    // Set CUDA device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // Parse command line arguments
    int block_dim_x = atoi(argv[1]);
    int block_dim_y = atoi(argv[2]);

    int size = (hN + 2) * (hN + 2);
    int nBytes = size * sizeof(float);
    double iStart, iElaps;

    // Allocate host memory
    float *u, *u_prev, *v, *v_prev, *dens, *dens_prev;
    u = (float*)malloc(nBytes);
    u_prev = (float*)malloc(nBytes);
    v = (float*)malloc(nBytes);
    v_prev = (float*)malloc(nBytes);
    dens = (float*)malloc(nBytes);
    dens_prev = (float*)malloc(nBytes);
    
    // Allocate device memory
    float *d_u, *d_u_prev, *d_v, *d_v_prev, *d_dens, *d_dens_prev, *d_uTemp, *d_vTemp, *d_densTemp;
    CHECK(hipMalloc((void **)&d_u, nBytes));
    CHECK(hipMalloc((void **)&d_u_prev, nBytes));
    CHECK(hipMalloc((void **)&d_v, nBytes));
    CHECK(hipMalloc((void **)&d_v_prev, nBytes));
    CHECK(hipMalloc((void **)&d_dens, nBytes));
    CHECK(hipMalloc((void **)&d_dens_prev, nBytes));
    CHECK(hipMalloc((void **)&d_uTemp, nBytes));
    CHECK(hipMalloc((void **)&d_vTemp, nBytes));
    CHECK(hipMalloc((void **)&d_densTemp, nBytes));

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(N), &hN, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(DT), &hDT, sizeof(float)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(VIS), &hVIS, sizeof(float)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(DIFF), &hDIFF, sizeof(float)));

    // Define grid and block dimensions
    dim3 block, grid, nGrid;
    block = dim3(block_dim_x, block_dim_y);
    grid = dim3(((hN + 2) + block_dim_x - 1) / block_dim_x, ((hN + 2) + block_dim_y - 1) / block_dim_y);
    nGrid = dim3(grid.x / (1 << GRID_DIVISION_FACTOR), grid.y / (1 << GRID_DIVISION_FACTOR));

    // Simulation
    int z = 0;
    int first = 1;
    iStart = cpuSecond();
    while (z++ < 1) {
        if (first) {
            initializeParameters(dens, dens_prev, u, u_prev, v, v_prev);
            first = 0;
            //printf("HELLO init\n");
            //printStateGrid(dens_prev, u_prev, v_prev);
            //printf("Hello end\n");

            CHECK(hipMemcpy(d_u, u, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_u_prev, u_prev, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_v, v, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_v_prev, v_prev, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_dens, dens, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_dens_prev, dens_prev, nBytes, hipMemcpyHostToDevice));
        } else {
            for (int i = 0; i < size; i++) {
                u_prev[i] = 0.0f;
                v_prev[i] = 0.0f;
                dens_prev[i] = 0.0f;
            }

            CHECK(hipMemcpy(d_u_prev, u_prev, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_v_prev, v_prev, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_dens_prev, dens_prev, nBytes, hipMemcpyHostToDevice));
        }

        vel_step(nGrid, block, d_u, d_v, d_u_prev, d_v_prev, d_uTemp, d_vTemp);
        dens_step(nGrid, block, d_dens, d_dens_prev, d_u, d_v, d_densTemp);

        // DA METTERE NEL CICLO SOLO PER DEBUG 
        /*
        CHECK(cudaMemcpy(u, d_u, nBytes, cudaMemcpyDeviceToHost));
        CHECK(cudaMemcpy(u_prev, d_u_prev, nBytes, cudaMemcpyDeviceToHost));
        CHECK(cudaMemcpy(v, d_v, nBytes, cudaMemcpyDeviceToHost));
        CHECK(cudaMemcpy(v_prev, d_v_prev, nBytes, cudaMemcpyDeviceToHost));
        CHECK(cudaMemcpy(dens, d_dens, nBytes, cudaMemcpyDeviceToHost));
        CHECK(cudaMemcpy(dens_prev, d_dens_prev, nBytes, cudaMemcpyDeviceToHost));
        printStateGrid(dens, u, v);
        */     
    }
    iElaps = cpuSecond() - iStart;
    printf("grid: %d, <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", size, nGrid.x, nGrid.y, block.x, block.y, iElaps);

    CHECK(hipMemcpy(u, d_u, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(v, d_v, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(dens, d_dens, nBytes, hipMemcpyDeviceToHost));
    //printStateGrid(dens, u, v);        

    // Cleaning
    free(u); free(u_prev);
    free(v); free(v_prev);
    free(dens); free(dens_prev);

    CHECK(hipFree(d_u)); CHECK(hipFree(d_u_prev));
    CHECK(hipFree(d_v)); CHECK(hipFree(d_v_prev));
    CHECK(hipFree(d_dens)); CHECK(hipFree(d_dens_prev));
    CHECK(hipFree(d_uTemp)); CHECK(hipFree(d_vTemp)); CHECK(hipFree(d_densTemp));

    return 0;
}