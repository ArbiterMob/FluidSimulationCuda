// CUDA version - 1 cell for each thread + Shared Memory

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

#include <hip/hip_runtime.h>

// GPU Constants
__constant__ int N;       // (WidthGrid - 2) 
__constant__ float DT;    // Instantaneous change in time (timestep)
__constant__ float VIS;   // Viscosity coefficient
__constant__ float DIFF;  // Diffusion coefficient

// CPU Global Variables
int hN = (1<<3) - 2;
float hDT = 0.016f;
float hVIS = 0.0025f;
float hDIFF = 0.1f;

// SWAP macro
#define SWAP(x0, x) {float *tmp = x0; x0 = x; x = tmp;}

// Function to check CUDA errors
#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if (error != hipSuccess) \
    { \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
        fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(EXIT_FAILURE); \
    } \
}

// Function to debug a float grid
void printDebug(/*char *string,*/ float *x) {
    int i, j;

    //printf("Debugging -> %s\n", string);

    for (i = 0; i < hN + 2; i++) {
        for (j = 0; j < hN + 2; j++) {
            printf("[%f] ", x[j + i * (hN + 2)]);
        }
        printf("\n");
    }
    printf("\n\n");
}

// Function to debug and print the state of the grid
void printStateGrid(float *dens, float *u, float *v) {
    int i, j;

    printf("---------------------------------------\n");
    printf("DENSITY\n");
    for (i = 0; i < hN + 2; i++) {
        for (j = 0; j < hN + 2; j++) {
            printf("[%f] ", dens[j + i * (hN +2)]);
        }
        printf("\n");
    }
    printf("\n\n");

    printf("VELOCITY\n");
    for (i = 0; i < hN + 2; i++) {
        for (j = 0; j < hN + 2; j++) {
            printf("[%f, %f] ", u[j + i * (hN +2)], v[j + i * (hN +2)]);
        }
        printf("\n");
    }
}

// Function to measure time in seconds
double cpuSecond() {
    struct timespec ts;
    timespec_get(&ts, TIME_UTC);
    return ((double)ts.tv_sec + (double)ts.tv_nsec * 1.e-9);
}

// CUDA device function to set the boundary conditions (borders)
__device__ void set_bndOnGPU(int b, float *d_x, int ix, int iy, int tid) {
    if (ix == 0 && iy < N + 1 && iy > 0)
        d_x[tid] = b == 1 ? -d_x[tid + 1] : d_x[tid + 1];
    else if (ix == N + 1 && iy < N + 1 && iy > 0)
        d_x[tid] = b == 1 ? -d_x[tid - 1] : d_x[tid - 1];
    else if (iy == 0 && ix > 0 && ix < N + 1)
        d_x[tid] = b == 2 ? -d_x[tid + N + 2] : d_x[tid + N + 2];
    else if (iy == N + 1 && ix > 0 && ix < N + 1)
        d_x[tid] = b == 2 ? -d_x[tid - N - 2] : d_x[tid - N - 2];
}   

// CUDA device function to set the boundary conditions (corners)
__device__ void set_crnOnGPU(int b, float *d_x, int ix, int iy, int tid) {
    if (ix == 0 && iy == 0)
        d_x[tid] = 0.5f * (d_x[tid + 1] + d_x[tid + N + 2]);
    else if (ix == 0 && iy == N + 1)
        d_x[tid] = 0.5f * (d_x[tid + 1] + d_x[tid - N - 2]);
    else if (ix == N + 1 && iy == 0)
        d_x[tid] = 0.5f * (d_x[tid - 1] + d_x[tid + N + 2]);
    else if (ix == N + 1 && iy == N + 1) 
        d_x[tid] = 0.5f * (d_x[tid - 1] + d_x[tid - N - 2]);
}

// CUDA kernel function to  ADD EXTERNAL SOURCES
__global__ void add_sourceOnGPU(float *d_x, float *d_s) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int tid = ix + iy * (N + 2);
    int sizeShared = blockDim.x * blockDim.y;
    
    extern __shared__ float array[];
    
    float *d_x_s = (float*)array;
    float *d_s_s = (float*)&d_x_s[sizeShared];


    int sTid = threadIdx.x + threadIdx.y * blockDim.x;

    if (ix < (N + 2) && iy < (N + 2)) {
        d_x_s[sTid] = d_x[tid];
       // __syncthreads(); // to avoid bank conflicts?
        d_s_s[sTid] = d_s[tid];
        __syncthreads();

        //printf("sTid %d | tid %d | d_x_s[sTid] %f | d_s_s[sTid] %f, d_x[tid] %f | d_s[tid] %f\n",
        //    sTid, tid, d_x_s[sTid], d_s_s[sTid], d_x[tid], d_s[tid]);
        d_x_s[sTid] += DT * d_s_s[sTid];

        d_x[tid] = d_x_s[sTid];
        //__syncthreads(); // to avoid bank conflicts?
        d_s[tid] = d_s_s[sTid];

    }
}

// CUDA kernel function to perform DIFFUSION (using Jacobi iteration outside of kernel)
__global__ void diffuseOnGPU(int b, float *d_x, float *d_x0, float *d_xTemp, float alpha, float beta) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int tid = ix + iy * (N + 2);
    int sizeShared = (blockDim.x) * (blockDim.y);
    
    extern __shared__ float array[];

    float *d_x_s = (float*)array;
    float *d_x0_s = (float*)&array[sizeShared];
    float *d_xTemp_s = (float*)&array[sizeShared * 2];

    int sTid = threadIdx.x + threadIdx.y * blockDim.x;

    if (ix < (N + 2) && iy < (N + 2)) {
        d_x_s[sTid] = d_x[tid];
        d_x0_s[sTid] = d_x0[tid];

        if (ix >= 1 && ix <= N && iy >= 1 && iy <= N) {
            d_xTemp_s[sTid] = (d_x0_s[sTid] + alpha * (d_x_s[sTid - 1] + d_x_s[sTid + 1] + 
                d_x_s[sTid - N - 2] + d_x_s[sTid + N + 2])) / beta;
            //printf("d_xTemp_s id %f in (%d, %d): d_x0_s[sTid] %f | d_x_s[sTid - 1] %f | d_x_s[sTid + 1] %f, d_x_s[sTid - N - 2] %f | d_x_s[sTid + N + 2] %f | beta %f\n",
            //    d_xTemp_s[sTid], ix, iy, d_x0_s[sTid], d_x_s[sTid - 1], d_x_s[sTid + 1], d_x_s[sTid - N - 2], d_x_s[sTid + N + 2], beta);
        }
    
        // Inefficient because multiple synchronization
        if (blockIdx.x == 0 || blockIdx.x == gridDim.x - 1 || blockIdx.y == 0 || blockIdx.y == gridDim.y - 1) {
            __syncthreads(); // synchronization intra-block to ensure that the 
                         // threads in the boundary can compute correctly
            set_bndOnGPU(b, d_xTemp, ix, iy, tid);

            __syncthreads(); // now that all the borders are complete, we can compute
                         // corners

            set_crnOnGPU(b, d_xTemp, ix, iy, tid);
        }
    }
}

// Function to simulate the evolution of density
void dens_step(dim3 grid, dim3 block, float *d_x, float *d_x0, float *d_u, float *d_v, float *d_densTemp) {
    int nBytes = sizeof(float) * block.x * block.y;
    
    add_sourceOnGPU<<<grid, block, 2 * nBytes>>>(d_x, d_x0);

    SWAP(d_x0, d_x);
}

// Function to simulate the evolution of velocity
void vel_step(dim3 grid, dim3 block, float *d_u, float *d_v, float *d_u0, float *d_v0, float *d_uTemp, float *d_vTemp) {
    int nBytes = sizeof(float) * block.x * block.y;
    
    add_sourceOnGPU<<<grid, block, 2 * nBytes>>>(d_u, d_u0);
    add_sourceOnGPU<<<grid, block>>>(d_v, d_v0);

    hipDeviceSynchronize();

    SWAP(d_u0, d_u);
    SWAP(d_v0, d_v);

    /*
    cudaDeviceSynchronize();
    float *temp = (float*)malloc(sizeof(float) * (hN + 2) * (hN + 2));
    cudaMemcpy(temp, d_u0, sizeof(float) * (hN + 2) * (hN + 2), cudaMemcpyDeviceToHost);
    printf("u0\n");
    printDebug(temp);
    */

    float alpha = hDT * hVIS * hN * hN;
    float beta = 1 + 4 * alpha;
}

// Function to initialize the density and velocity
void initializeParameters(float *dens, float *dens_prev, float *u, float *u_prev, float *v, float *v_prev) {
    int i, j;
    int center_x = (hN + 2) / 2, center_y = (hN + 2) / 2;
    int radius = (hN + 2) / 8;

    // density source
    for (i = 0; i < hN + 2; i++) {
        for (j = 0; j < hN + 2; j++) {
            if ((j < center_x + radius) && (j >= center_x - radius) &&
                (i < center_y + radius) && (i >= center_y - radius))
                dens_prev[j + i * (hN + 2)] = (rand() % 100) / 1000.0f;
            else 
                dens_prev[j + i * (hN + 2)] = 0.0f;
            dens[j + i * (hN + 2)] = 0.0f;
        }
    } 

    // velocity
    for (i = 0; i < hN + 2; i++) {
        for (j = 0; j < hN + 2; j++) {
            u_prev[j + i * (hN +2)] = (rand() % 100) / 100.0f;
            v_prev[j + i * (hN +2)] = (rand() % 100) / 100.0f;

            u[j + i * (hN +2)] = 0.0f;
            v[j + i * (hN +2)] = 0.0f;
        }
    }
}

int main(int argc, char **argv) {
     // Check command line arguments
    if (argc != 3) {
        printf("Usage: %s <block_dim_x> <block_dim_y>\n", argv[0]);
        return 1;
    }
    
    // Set CUDA device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // Parse command line arguments
    int block_dim_x = atoi(argv[1]);
    int block_dim_y = atoi(argv[2]);

    int size = (hN + 2) * (hN + 2);
    int nBytes = size * sizeof(float);
    double iStart, iElaps;

    // Allocate host memory
    float *u, *u_prev, *v, *v_prev, *dens, *dens_prev;
    u = (float*)malloc(nBytes);
    u_prev = (float*)malloc(nBytes);
    v = (float*)malloc(nBytes);
    v_prev = (float*)malloc(nBytes);
    dens = (float*)malloc(nBytes);
    dens_prev = (float*)malloc(nBytes);
    
    // Allocate device memory
    float *d_u, *d_u_prev, *d_v, *d_v_prev, *d_dens, *d_dens_prev, *d_uTemp, *d_vTemp, *d_densTemp;
    CHECK(hipMalloc((void **)&d_u, nBytes));
    CHECK(hipMalloc((void **)&d_u_prev, nBytes));
    CHECK(hipMalloc((void **)&d_v, nBytes));
    CHECK(hipMalloc((void **)&d_v_prev, nBytes));
    CHECK(hipMalloc((void **)&d_dens, nBytes));
    CHECK(hipMalloc((void **)&d_dens_prev, nBytes));
    CHECK(hipMalloc((void **)&d_uTemp, nBytes));
    CHECK(hipMalloc((void **)&d_vTemp, nBytes));
    CHECK(hipMalloc((void **)&d_densTemp, nBytes));

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(N), &hN, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(DT), &hDT, sizeof(float)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(VIS), &hVIS, sizeof(float)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(DIFF), &hDIFF, sizeof(float)));

    // Define grid and block dimensions
    dim3 block, grid;
    block = dim3(block_dim_x, block_dim_y);
    grid = dim3(((hN + 2) + block_dim_x - 1) / block_dim_x, ((hN + 2) + block_dim_y - 1) / block_dim_y);

    // Simulation
    int z = 0;
    int first = 1;
    iStart = cpuSecond();
    while (z++ < 1) {
        if (first) {
            initializeParameters(dens, dens_prev, u, u_prev, v, v_prev);
            first = 0;
            //printf("HELLO init\n");
            //printStateGrid(dens_prev, u_prev, v_prev);
            //printf("Hello end\n");

            CHECK(hipMemcpy(d_u, u, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_u_prev, u_prev, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_v, v, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_v_prev, v_prev, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_dens, dens, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_dens_prev, dens_prev, nBytes, hipMemcpyHostToDevice));
        } else {
            for (int i = 0; i < size; i++) {
                u_prev[i] = 0.0f;
                v_prev[i] = 0.0f;
                dens_prev[i] = 0.0f;
            }

            CHECK(hipMemcpy(d_u_prev, u_prev, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_v_prev, v_prev, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_dens_prev, dens_prev, nBytes, hipMemcpyHostToDevice));
        }

        vel_step(grid, block, d_u, d_v, d_u_prev, d_v_prev, d_uTemp, d_vTemp);
        dens_step(grid, block, d_dens, d_dens_prev, d_u, d_v, d_densTemp);

        // DA METTERE NEL CICLO SOLO PER DEBUG 
        /*
        CHECK(cudaMemcpy(u, d_u, nBytes, cudaMemcpyDeviceToHost));
        CHECK(cudaMemcpy(u_prev, d_u_prev, nBytes, cudaMemcpyDeviceToHost));
        CHECK(cudaMemcpy(v, d_v, nBytes, cudaMemcpyDeviceToHost));
        CHECK(cudaMemcpy(v_prev, d_v_prev, nBytes, cudaMemcpyDeviceToHost));
        CHECK(cudaMemcpy(dens, d_dens, nBytes, cudaMemcpyDeviceToHost));
        CHECK(cudaMemcpy(dens_prev, d_dens_prev, nBytes, cudaMemcpyDeviceToHost));
        printStateGrid(dens, u, v);
        */     
    }
    CHECK(hipDeviceSynchronize());
    iElaps = cpuSecond() - iStart;
    printf("grid: %d, <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", size, grid.x, grid.y, block.x, block.y, iElaps);

    CHECK(hipMemcpy(u, d_u, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(v, d_v, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(dens, d_dens, nBytes, hipMemcpyDeviceToHost));
    printStateGrid(dens, u, v);        

    // Cleaning
    free(u); free(u_prev);
    free(v); free(v_prev);
    free(dens); free(dens_prev);

    CHECK(hipFree(d_u)); CHECK(hipFree(d_u_prev));
    CHECK(hipFree(d_v)); CHECK(hipFree(d_v_prev));
    CHECK(hipFree(d_dens)); CHECK(hipFree(d_dens_prev));
    CHECK(hipFree(d_uTemp)); CHECK(hipFree(d_vTemp)); CHECK(hipFree(d_densTemp));

    return 0;
}