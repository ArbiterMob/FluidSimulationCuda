#include "hip/hip_runtime.h"
// CUDA first version naive (multiple cells for each thread (strided))

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

// GPU Constants
__constant__ int N;       // (WidthGrid - 2) 
__constant__ float DT;    // Instantaneous change in time (timestep)
__constant__ float VIS;   // Viscosity coefficient
__constant__ float DIFF;  // Diffusion coefficient

// CPU Global Variables
int hN = (1<<7) - 2;
float hDT = 0.016f;
float hVIS = 0.0025f;
float hDIFF = 0.1f;
int GRID_DIVISION_FACTOR = 4; // explain what this is !!!

namespace cg = cooperative_groups;

// SWAP macro
#define SWAP(x0, x) {float *tmp = x0; x0 = x; x = tmp;}

// Function to check CUDA errors
#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if (error != hipSuccess) \
    { \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
        fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(EXIT_FAILURE); \
    } \
}

// Function to debug a float grid
void printDebug(/*char *string,*/ float *x) {
    int i, j;

    //printf("Debugging -> %s\n", string);

    for (i = 0; i < hN + 2; i++) {
        for (j = 0; j < hN + 2; j++) {
            printf("[%f] ", x[j + i * (hN + 2)]);
        }
        printf("\n");
    }
    printf("\n\n");
}

// Function to debug and print the state of the grid
void printStateGrid(float *dens, float *u, float *v) {
    int i, j;

    printf("---------------------------------------\n");
    printf("DENSITY\n");
    for (i = 0; i < hN + 2; i++) {
        for (j = 0; j < hN + 2; j++) {
            printf("[%f] ", dens[j + i * (hN +2)]);
        }
        printf("\n");
    }
    printf("\n\n");

    printf("VELOCITY\n");
    for (i = 0; i < hN + 2; i++) {
        for (j = 0; j < hN + 2; j++) {
            printf("[%f, %f] ", u[j + i * (hN +2)], v[j + i * (hN +2)]);
        }
        printf("\n");
    }
}

// Function to measure time in seconds
double cpuSecond() {
    struct timespec ts;
    timespec_get(&ts, TIME_UTC);
    return ((double)ts.tv_sec + (double)ts.tv_nsec * 1.e-9);
}

// CUDA device function to set the boundary conditions (borders)
__device__ void set_bndOnGPU(int b, float *d_x, int ix, int iy, int tid) {
    if (ix == 0 && iy < N + 1 && iy > 0)
        d_x[tid] = b == 1 ? -d_x[tid + 1] : d_x[tid + 1];
    else if (ix == N + 1 && iy < N + 1 && iy > 0)
        d_x[tid] = b == 1 ? -d_x[tid - 1] : d_x[tid - 1];
    else if (iy == 0 && ix > 0 && ix < N + 1)
        d_x[tid] = b == 2 ? -d_x[tid + N + 2] : d_x[tid + N + 2];
    else if (iy == N + 1 && ix > 0 && ix < N + 1)
        d_x[tid] = b == 2 ? -d_x[tid - N - 2] : d_x[tid - N - 2];
}   

// CUDA device function to set the boundary conditions (corners)
__device__ void set_crnOnGPU(int b, float *d_x, int ix, int iy, int tid) {
    if (ix == 0 && iy == 0)
        d_x[tid] = 0.5f * (d_x[tid + 1] + d_x[tid + N + 2]);
    else if (ix == 0 && iy == N + 1)
        d_x[tid] = 0.5f * (d_x[tid + 1] + d_x[tid - N - 2]);
    else if (ix == N + 1 && iy == 0)
        d_x[tid] = 0.5f * (d_x[tid - 1] + d_x[tid + N + 2]);
    else if (ix == N + 1 && iy == N + 1) 
        d_x[tid] = 0.5f * (d_x[tid - 1] + d_x[tid - N - 2]);
}

__global__ void boundaryGPU(int b, float *d_x) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (blockIdx.x == 0) {
        set_bndOnGPU(b, d_x, ix, 0, ix);
        set_bndOnGPU(b, d_x, ix, N + 1, ix + (N + 1) * (N + 2));
        set_bndOnGPU(b, d_x, 0, ix, ix * (N + 2));
        set_bndOnGPU(b, d_x, 0, N + 1 - ix, (N + 1 - ix) * (N + 2));
        __syncthreads();
        set_crnOnGPU(b, d_x, 0, 0, 0);
        set_crnOnGPU(b, d_x, 0, N + 1, (N + 1) * (N + 2));
    } else if (blockIdx.x > 0 && blockIdx.x < gridDim.x - 1) {
        set_bndOnGPU(b, d_x, ix, 0, ix);
        set_bndOnGPU(b, d_x, 0, ix, ix * (N + 2));
        set_bndOnGPU(b, d_x, N + 1, ix, N + 1 + ix * (N + 2));
        set_bndOnGPU(b, d_x, ix, N + 1, ix + (N + 1) * (N + 2));
    } else if (blockIdx.x == gridDim.x - 1) {
        set_bndOnGPU(b, d_x, ix, 0, ix);
        set_bndOnGPU(b, d_x, ix, N + 1, ix + (N + 1) * (N + 2));
        set_bndOnGPU(b, d_x, N + 1, ix, N + 1 + ix * (N + 2));
        set_bndOnGPU(b, d_x, N + 1, N + 1 - ix, N + 1 + (N + 1 - ix) * (N + 2));
        __syncthreads();
        set_crnOnGPU(b, d_x, N + 1, 0, N + 1);
        set_crnOnGPU(b, d_x, N + 1, N + 1, (N + 1) + (N + 1) * (N + 2));
    }
}

// CUDA kernel function to  ADD EXTERNAL SOURCES
__global__ void add_sourceOnGPU(float *d_x, float *d_s) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int size = N + 2;
    int i, j;

    for (i = iy; i < size; i += blockDim.y * gridDim.y) {
        for (j = ix; j < size; j += blockDim.x * gridDim.x) {
            int tid = j + i * (N + 2);
            if (j < (N + 2) && i < (N + 2))
                d_x[tid] += DT * d_s[tid];
        }
    }
}

// CUDA kernel function to perform DIFFUSION (using Jacobi iteration outside of kernel)
__global__ void diffuseOnGPU(int b, float *d_x, float *d_x0, float *d_xTemp, float alpha, float beta) {
    cg::grid_group grid = cg::this_grid();
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int size = N + 2;
    int i, j;

    for (i = iy; i < size; i += blockDim.y * gridDim.y) {
        for (j = ix; j < size; j += blockDim.x * gridDim.x) {
            int tid = j + i * (N + 2);
            if (j >= 1 && j <= N && i >= 1 && i <= N) {
                d_xTemp[tid] = (d_x0[tid] + alpha * (d_x[tid - 1] + d_x[tid + 1] + 
                    d_x[tid - N - 2] + d_x[tid + N + 2])) / beta;
            }
        }
    }
    /*
    __syncthreads();

    // Inefficient because multiple synchronization
    if (blockIdx.x == 0 || blockIdx.x == gridDim.x - 1 || blockIdx.y == 0 || blockIdx.y == gridDim.y - 1 &&
        ix < N + 2 && iy < N + 2) {
        //__syncthreads(); // synchronization intra-block to ensure that the 
                     // threads in the boundary can compute correctly

        for (i = iy; i < size; i += blockDim.y * gridDim.y) {
            for (j = ix; j < size; j += blockDim.x * gridDim.x) {
                int tid = j + i * (N + 2);
                set_bndOnGPU(b, d_xTemp, j, i, tid);
            }
        }
    }

    __syncthreads(); // now that all the borders are complete, we can compute
                         // corners

    if (blockIdx.x == 0 || blockIdx.x == gridDim.x - 1 || blockIdx.y == 0 || blockIdx.y == gridDim.y - 1 &&
        ix < N + 2 && iy < N + 2) {
        for (i = iy; i < size; i += blockDim.y * gridDim.y) {
            for (j = ix; j < size; j += blockDim.x * gridDim.x) {
                int tid = j + i * (N + 2);
                set_crnOnGPU(b, d_xTemp, j, i, tid);
            }
        }
    }
    */
}

// CUDA kernel function to perform ADVECTION (using bilinear interpolation)
__global__ void advectOnGPU(int b, float *d_d, float *d_d0, float *d_u, float *d_v) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int size = N + 2;
    int i, j;

    int i0, j0, i1, j1;
    float x, y, s0, t0, s1, t1, dt0;

    dt0 = DT * N;
    for (i = iy; i < size; i += blockDim.y * gridDim.y) {
        for (j = ix; j < size; j += blockDim.x * gridDim.x) {
            int tid = j + i * (N + 2);
            if (j >= 1 && j <= N && i >= 1 && i <= N) {
                x = j - dt0 * d_u[tid];
                y = i - dt0 * d_v[tid];

                if (x < 0.5) 
                    x = 0.5; 
                if (x > N + 0.5) 
                    x = N + 0.5;
                j0 = (int)x;
                j1 = j0 + 1;

                if (y < 0.5) 
                    y = 0.5; 
                if (y > N + 0.5) 
                    y = N + 0.5; 
                i0 = (int)y;
                i1 = i0 + 1;

                s1 = x - j0;
                s0 = 1 - s1;
                t1 = y - i0;
                t0 = 1 - t1;

                d_d[tid] = s0 * (t0 * d_d0[j0 + i0 * (N + 2)] + t1 * d_d0[j0 + i1 * (N + 2)]) +
                    s1 * (t0 * d_d0[j1 + i0 * (N + 2)] + t1 * d_d0[j1 + i1 * (N + 2)]);
                //printf("(%f, %f) | d_d[tid] %f (%d, %d) | d_d0[j0 + i0 * (N + 2)] %f | d_d0[j0 + i1 * (N + 2)] %f | d_d0[j1 + i0 * (N + 2)] %f | d_d0[j1 + i1 * (N + 2)] %f\n",
                //    x, y, d_d[tid], ix, iy, d_d0[j0 + i0 * (N + 2)], d_d0[j0 + i1 * (N + 2)], d_d0[j1 + i0 * (N + 2)], d_d0[j1 + i1 * (N + 2)]);
            }
        }
    }

    /*
    __syncthreads();

    // Inefficient because multiple synchronization
    if (blockIdx.x == 0 || blockIdx.x == gridDim.x - 1 || blockIdx.y == 0 || blockIdx.y == gridDim.y - 1 &&
        ix < N + 2 && iy < N + 2) {
        //__syncthreads(); // synchronization intra-block to ensure that the 
                     // threads in the boundary can compute correctly

        for (i = iy; i < size; i += blockDim.y * gridDim.y) {
            for (j = ix; j < size; j += blockDim.x * gridDim.x) {
                int tid = j + i * (N + 2);
                set_bndOnGPU(b, d_d, j, i, tid);
            }
        }
    }

    __syncthreads();

    if (blockIdx.x == 0 || blockIdx.x == gridDim.x - 1 || blockIdx.y == 0 || blockIdx.y == gridDim.y - 1 &&
        ix < N + 2 && iy < N + 2) {
        //__syncthreads(); // now that all the borders are complete, we can compute
                         // corners

        for (i = iy; i < size; i += blockDim.y * gridDim.y) {
            for (j = ix; j < size; j += blockDim.x * gridDim.x) {
                int tid = j + i * (N + 2);
                set_crnOnGPU(b, d_d, j, i, tid);
            }
        }
    }
    */
}

// CUDA kernel function to COMPUTE DIVERGENCE AND PRESSURE
__global__ void computeDivergenceAndPressureOnGPU(float *d_u, float *d_v, float *p, float *div) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int size = N + 2;
    int i, j;

    float h = 1.0f / N;
    for (i = iy; i < size; i += blockDim.y * gridDim.y) {
        for (j = ix; j < size; j += blockDim.x * gridDim.x) {
            int tid = j + i * (N + 2);
            if (j >= 1 && j <= N && i >= 1 && i <= N) {
                div[tid] = -0.5f * h * (d_u[tid + 1] - d_u[tid - 1] + d_v[tid + N + 2] - d_v[tid - N - 2]);
                p[tid] = 0.0f;
                //printf("div[tid] %f (%d, %d)| d_u[tid + 1] %f | d_u[tid - 1] %f | d_v[tid + N + 2] %f | d_v[tid - N - 2] %f\n",
                //    div[tid], ix, iy, d_u[tid + 1], d_u[tid - 1], d_v[tid + N + 2], d_v[tid - N - 2]);
            }
        }
    }

    /*
    __syncthreads();

    // Inefficient because multiple synchronization
    if (blockIdx.x == 0 || blockIdx.x == gridDim.x - 1 || blockIdx.y == 0 || blockIdx.y == gridDim.y - 1 &&
        ix < N + 2 && iy < N + 2) {
        //__syncthreads(); // synchronization intra-block to ensure that the 
                     // threads in the boundary can compute correctly

        for (i = iy; i < size; i += blockDim.y * gridDim.y) {
            for (j = ix; j < size; j += blockDim.x * gridDim.x) {
                int tid = j + i * (N + 2);
                set_bndOnGPU(0, div, j, i, tid);
                set_bndOnGPU(0, p, j, i, tid);
            }
        }
    }

    __syncthreads(); // now that all the borders are complete, we can compute
                         // corners

    if (blockIdx.x == 0 || blockIdx.x == gridDim.x - 1 || blockIdx.y == 0 || blockIdx.y == gridDim.y - 1 &&
        ix < N + 2 && iy < N + 2) {
        for (i = iy; i < size; i += blockDim.y * gridDim.y) {
            for (j = ix; j < size; j += blockDim.x * gridDim.x) {
                int tid = j + i * (N + 2);
                set_crnOnGPU(0, div, j, i, tid);
                set_crnOnGPU(0, p, j, i, tid);
            }
        }
    }
    */
}

// CUDA kernel to perform the LAST PROJECTION STEP (using Jacobi iteration outside of kernel)
__global__ void lastProjectOnGPU(float *d_u, float *d_v, float *p) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int size = N + 2;
    int i, j;

    float h = 1.0f / N;
    for (i = iy; i < size; i += blockDim.y * gridDim.y) {
        for (j = ix; j < size; j += blockDim.x * gridDim.x) {
            int tid = j + i * (N + 2);
            if (j >= 1 && j <= N && i >= 1 && i <= N) {
                d_u[tid] -= 0.5f * (p[tid + 1] - p[tid - 1]) / h;
                d_v[tid] -= 0.5f * (p[tid + N + 2] - p[tid - N - 2]) / h;
            }
        }
    }

    /*
    __syncthreads();

    // Inefficient because multiple synchronization
    if (blockIdx.x == 0 || blockIdx.x == gridDim.x - 1 || blockIdx.y == 0 || blockIdx.y == gridDim.y - 1 &&
        ix < N + 2 && iy < N + 2) {
        //__syncthreads(); // synchronization intra-block to ensure that the 
                     // threads in the boundary can compute correctly

        for (i = iy; i < size; i += blockDim.y * gridDim.y) {
            for (j = ix; j < size; j += blockDim.x * gridDim.x) {
                int tid = j + i * (N + 2);
                set_bndOnGPU(1, d_u, j, i, tid);
                set_bndOnGPU(2, d_v, j, i, tid);
            }
        }
    }

    __syncthreads(); // now that all the borders are complete, we can compute
                     // corners

    if (blockIdx.x == 0 || blockIdx.x == gridDim.x - 1 || blockIdx.y == 0 || blockIdx.y == gridDim.y - 1 &&
        ix < N + 2 && iy < N + 2) {
        for (i = iy; i < size; i += blockDim.y * gridDim.y) {
            for (j = ix; j < size; j += blockDim.x * gridDim.x) {
                int tid = j + i * (N + 2);
                set_crnOnGPU(1, d_u, j, i, tid);
                set_crnOnGPU(2, d_v, j, i, tid);
            }
        }
    }
    */
}

// Function to simulate the evolution of density
void dens_step(dim3 grid, dim3 block, int boundGrid, float *d_x, float *d_x0, float *d_u, float *d_v, float *d_densTemp) {
    add_sourceOnGPU<<<grid, block>>>(d_x, d_x0);

    float alpha = hDT * hDIFF * hN * hN;
    float beta = 1 + 4 * alpha;
    SWAP(d_x0, d_x);
    for (int k = 0; k < 40; k++) { // inefficient -> multiple kernel calls
        diffuseOnGPU<<<grid, block>>>(0, d_x, d_x0, d_densTemp, alpha, beta);
        boundaryGPU<<<boundGrid, block.x>>>(0, d_densTemp);
        SWAP(d_densTemp, d_x);
    }
    
    SWAP(d_x0, d_x);
    advectOnGPU<<<grid, block>>>(0, d_x, d_x0, d_u, d_v);
    boundaryGPU<<<boundGrid, block.x>>>(0, d_x);
}

// Function to simulate the evolution of velocity
void vel_step(dim3 grid, dim3 block, int boundGrid, float *d_u, float *d_v, float *d_u0, float *d_v0, float *d_uTemp, float *d_vTemp) {
    add_sourceOnGPU<<<grid, block>>>(d_u, d_u0);
    add_sourceOnGPU<<<grid, block>>>(d_v, d_v0);

    SWAP(d_u, d_u0);
    SWAP(d_v, d_v0);

    float alpha = hDT * hVIS * hN * hN;
    float beta = 1 + 4 * alpha;
    for (int k = 0; k < 40; k++) { // inefficient -> multiple kernel calls
        diffuseOnGPU<<<grid, block>>>(1, d_u, d_u0, d_uTemp, alpha, beta);
        diffuseOnGPU<<<grid, block>>>(2, d_v, d_v0, d_vTemp, alpha, beta);
        boundaryGPU<<<boundGrid, block.x>>>(1, d_uTemp);
        boundaryGPU<<<boundGrid, block.x>>>(2, d_vTemp);
        SWAP(d_uTemp, d_u);
        SWAP(d_vTemp, d_v);
    }
    
    computeDivergenceAndPressureOnGPU<<<grid, block>>>(d_u, d_v, d_u0, d_v0);
    boundaryGPU<<<boundGrid, block.x>>>(0, d_u0);
    boundaryGPU<<<boundGrid, block.x>>>(0, d_v0);

    alpha = 1;
    beta = 4;
    // d_u0 is p, d_v0 is div
    for (int k = 0; k < 40; k++) { // inefficient -> multiple kernel calls
        diffuseOnGPU<<<grid, block>>>(0, d_u0, d_v0, d_uTemp, alpha, beta);
        boundaryGPU<<<boundGrid, block.x>>>(0, d_uTemp);
        SWAP(d_uTemp, d_u0);
    }
    lastProjectOnGPU<<<grid, block>>>(d_u, d_v, d_u0);
    boundaryGPU<<<boundGrid, block.x>>>(1, d_u);
    boundaryGPU<<<boundGrid, block.x>>>(2, d_v);

    SWAP(d_u0, d_u);
    SWAP(d_v0, d_v);
    advectOnGPU<<<grid, block>>>(1, d_u, d_u0, d_u0, d_v0);
    advectOnGPU<<<grid, block>>>(2, d_v, d_v0, d_u0, d_v0);
    boundaryGPU<<<boundGrid, block.x>>>(1, d_u);
    boundaryGPU<<<boundGrid, block.x>>>(2, d_v);

    computeDivergenceAndPressureOnGPU<<<grid, block>>>(d_u, d_v, d_u0, d_v0);
    boundaryGPU<<<boundGrid, block.x>>>(0, d_u0);
    boundaryGPU<<<boundGrid, block.x>>>(0, d_v0);
    // d_u0 is p, d_v0 is div
    for (int k = 0; k < 40; k++) { // inefficient -> multiple kernel calls
        diffuseOnGPU<<<grid, block>>>(0, d_u0, d_v0, d_uTemp, alpha, beta);
        boundaryGPU<<<boundGrid, block.x>>>(0, d_uTemp);
        SWAP(d_uTemp, d_u0);
    }
    lastProjectOnGPU<<<grid, block>>>(d_u, d_v, d_u0);
    boundaryGPU<<<boundGrid, block.x>>>(1, d_u);
    boundaryGPU<<<boundGrid, block.x>>>(2, d_v);
}

// Function to initialize the density and velocity
void initializeParameters(float *dens, float *dens_prev, float *u, float *u_prev, float *v, float *v_prev) {
    int i, j;
    int center_x = (hN + 2) / 2, center_y = (hN + 2) / 2;
    int radius = (hN + 2) / 8;

    // density source
    for (i = 0; i < hN + 2; i++) {
        for (j = 0; j < hN + 2; j++) {
            if ((j < center_x + radius) && (j >= center_x - radius) &&
                (i < center_y + radius) && (i >= center_y - radius))
                dens_prev[j + i * (hN + 2)] = (rand() % 100) / 1000.0f;
            else 
                dens_prev[j + i * (hN + 2)] = 0.0f;
            dens[j + i * (hN + 2)] = 0.0f;
        }
    } 

    // velocity
    for (i = 0; i < hN + 2; i++) {
        for (j = 0; j < hN + 2; j++) {
            u_prev[j + i * (hN +2)] = (rand() % 100) / 100.0f;
            v_prev[j + i * (hN +2)] = (rand() % 100) / 100.0f;

            u[j + i * (hN +2)] = 0.0f;
            v[j + i * (hN +2)] = 0.0f;
        }
    }
}

int main(int argc, char **argv) {
     // Check command line arguments
    if (argc != 3) {
        printf("Usage: %s <block_dim_x> <block_dim_y>\n", argv[0]);
        return 1;
    }
    
    // Set CUDA device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // Parse command line arguments
    int block_dim_x = atoi(argv[1]);
    int block_dim_y = atoi(argv[2]);

    int size = (hN + 2) * (hN + 2);
    int nBytes = size * sizeof(float);
    double iStart, iElaps;

    // Allocate host memory
    float *u, *u_prev, *v, *v_prev, *dens, *dens_prev;
    u = (float*)malloc(nBytes);
    u_prev = (float*)malloc(nBytes);
    v = (float*)malloc(nBytes);
    v_prev = (float*)malloc(nBytes);
    dens = (float*)malloc(nBytes);
    dens_prev = (float*)malloc(nBytes);
    
    // Allocate device memory
    float *d_u, *d_u_prev, *d_v, *d_v_prev, *d_dens, *d_dens_prev, *d_uTemp, *d_vTemp, *d_densTemp;
    CHECK(hipMalloc((void **)&d_u, nBytes));
    CHECK(hipMalloc((void **)&d_u_prev, nBytes));
    CHECK(hipMalloc((void **)&d_v, nBytes));
    CHECK(hipMalloc((void **)&d_v_prev, nBytes));
    CHECK(hipMalloc((void **)&d_dens, nBytes));
    CHECK(hipMalloc((void **)&d_dens_prev, nBytes));
    CHECK(hipMalloc((void **)&d_uTemp, nBytes));
    CHECK(hipMalloc((void **)&d_vTemp, nBytes));
    CHECK(hipMalloc((void **)&d_densTemp, nBytes));

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(N), &hN, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(DT), &hDT, sizeof(float)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(VIS), &hVIS, sizeof(float)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(DIFF), &hDIFF, sizeof(float)));

    // Define grid and block dimensions
    dim3 block, grid, nGrid;
    block = dim3(block_dim_x, block_dim_y);
    grid = dim3(((hN + 2) + block_dim_x - 1) / block_dim_x, ((hN + 2) + block_dim_y - 1) / block_dim_y);
    nGrid = dim3(grid.x / (1 << GRID_DIVISION_FACTOR), grid.y / (1 << GRID_DIVISION_FACTOR));

    // Simulation
    int z = 0;
    int first = 1;
    iStart = cpuSecond();
    while (z++ < 1) {
        if (first) {
            initializeParameters(dens, dens_prev, u, u_prev, v, v_prev);
            first = 0;
            //printf("HELLO init\n");
            //printStateGrid(dens_prev, u_prev, v_prev);
            //printf("Hello end\n");

            CHECK(hipMemcpy(d_u, u, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_u_prev, u_prev, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_v, v, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_v_prev, v_prev, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_dens, dens, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_dens_prev, dens_prev, nBytes, hipMemcpyHostToDevice));
        } else {
            for (int i = 0; i < size; i++) {
                u_prev[i] = 0.0f;
                v_prev[i] = 0.0f;
                dens_prev[i] = 0.0f;
            }

            CHECK(hipMemcpy(d_u_prev, u_prev, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_v_prev, v_prev, nBytes, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_dens_prev, dens_prev, nBytes, hipMemcpyHostToDevice));
        }

        vel_step(nGrid, block, grid.x, d_u, d_v, d_u_prev, d_v_prev, d_uTemp, d_vTemp);
        dens_step(nGrid, block, grid.x, d_dens, d_dens_prev, d_u, d_v, d_densTemp);

        // DA METTERE NEL CICLO SOLO PER DEBUG 
        /*
        CHECK(hipMemcpy(u, d_u, nBytes, hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(u_prev, d_u_prev, nBytes, hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(v, d_v, nBytes, hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(v_prev, d_v_prev, nBytes, hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(dens, d_dens, nBytes, hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(dens_prev, d_dens_prev, nBytes, hipMemcpyDeviceToHost));
        printStateGrid(dens, u, v);
        */     
    }
    CHECK(hipDeviceSynchronize());
    iElaps = cpuSecond() - iStart;
    printf("grid: %d, <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", size, nGrid.x, nGrid.y, block.x, block.y, iElaps);

    CHECK(hipMemcpy(u, d_u, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(v, d_v, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(dens, d_dens, nBytes, hipMemcpyDeviceToHost));
    //printStateGrid(dens, u, v);        

    // Cleaning
    free(u); free(u_prev);
    free(v); free(v_prev);
    free(dens); free(dens_prev);

    CHECK(hipFree(d_u)); CHECK(hipFree(d_u_prev));
    CHECK(hipFree(d_v)); CHECK(hipFree(d_v_prev));
    CHECK(hipFree(d_dens)); CHECK(hipFree(d_dens_prev));
    CHECK(hipFree(d_uTemp)); CHECK(hipFree(d_vTemp)); CHECK(hipFree(d_densTemp));

    return 0;
}